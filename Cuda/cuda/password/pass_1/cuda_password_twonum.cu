#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>

/****************************************************************************
 * 
 *
 * Compile with:
 *   nvcc -o cudapass_twonum cuda_password_twonum.cu
 * 
 * Dr Kevan Buckley, University of Wolverhampton, January 2018
 *****************************************************************************/
__device__ int is_a_match(char *attempt){
char plain_password1[] ="AA24";
char plain_password2[] ="RA33";
char plain_password3[] ="MI22";
char plain_password4[] ="TA44";

char *r = attempt;
char *m = attempt;
char *t = attempt;
char *a = attempt;
char *r1 = plain_password1;
char *r2 = plain_password2;
char *r3 = plain_password3;
char *r4 = plain_password4;

 while(*r ==*r1){
 	if(*r == '\0')
	{
		printf("password:%s\n", plain_password1);
		break;
	}
	r++;
	r1++;
}
while(*m ==*r2){
 	if(*m == '\0')
	{
		printf("password:%s\n", plain_password2);
		break;
	}
	m++;
	r2++;
}
while(*t ==*r3){
 	if(*t == '\0')
	{
		printf("password:%s\n", plain_password3);
		break;
	}
	t++;
	r3++;
}
while(*a ==*r4){
 	if(*a == '\0')
	{
		printf("password: %s\n", plain_password4);
		return 1;
	}
	a++;
	r4++;
}
return 0;
}

__global__ void kernel(){
char n1, n2;

char password[7];
password[6] ='\0';

int i = blockIdx.x +65;
int j = threadIdx.x+65;
char firstMatch =i;
char secondMatch =j;

password[0] =firstMatch;
password[1] =secondMatch;
	for(n1='0'; n1<='9'; n1++){
		for(n2='0'; n2<='9'; n2++){
			
					password[2] =n1;
					password[3] =n2;
					
	if(is_a_match(password)){
	}
	else{
	//printf("tried: %s\n",password);
	}
     }
    }
  }



int time_difference(struct timespec *start, struct timespec *finish,long long int *difference) {
  long long int ds =  finish->tv_sec - start->tv_sec; 
  long long int dn =  finish->tv_nsec - start->tv_nsec; 

  if(dn < 0 ) {
    ds--;
    dn += 1000000000; 
  } 
  *difference = ds * 1000000000 + dn;
  return !(*difference > 0);
}

	
int main() {

 struct timespec start, finish;   
  long long int time_elapsed;

  clock_gettime(CLOCK_MONOTONIC, &start);

  kernel<<<26,26>>>();
hipDeviceSynchronize();
  

 clock_gettime(CLOCK_MONOTONIC, &finish);
  time_difference(&start, &finish, &time_elapsed);
  printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed, 
         (time_elapsed/1.0e9)); 
return 0;
}
